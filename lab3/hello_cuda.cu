
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

using namespace std;

__global__ void kernel(int* tab, int elem_number) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int step = gridDim.x * blockDim.x;

    for (; i < elem_number; i += step) {
	tab[i] = 2 * tab[i];
    }
}

int main() {
    const int elem_number = 4096;
    int tab_cpu[elem_number];

    int* tab_gpu;
    hipError_t status;

    for (int i = 0; i < elem_number; i++) {
	tab_cpu[i] = i;
    }
    
    status = hipMalloc((void**)&tab_gpu, sizeof(int) * elem_number);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    status = hipMemcpy(tab_gpu, tab_cpu, sizeof(int) * elem_number, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    kernel<<<2, 256, 0>>>(tab_gpu, elem_number);

    status = hipMemcpy(tab_cpu, tab_gpu, sizeof(int) * elem_number, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    status = hipFree(tab_gpu);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    for (int i = 0; i < 10; i++) {
	cout << "Index " << i << " value " << tab_cpu[i] << endl;
    }
    cout << "Index 4095 value " << tab_cpu[4095] << endl;

    return 0;
}
