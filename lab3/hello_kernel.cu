
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void kernel() {
    
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.x;
    int c = gridDim.x;
    int d = gridDim.x * blockDim.x;

    printf("Hello World, my number: %d, block number: %d, blocks: %d, threads: %d\n", a, b, c, d);

}

int main() {
    kernel<<<2, 256, 0>>>();
    hipDeviceSynchronize();    
    return 0;
}
