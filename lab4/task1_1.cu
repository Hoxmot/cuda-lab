#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

using namespace std;

#define LEN 10

__global__ void add_vec(int *v1, int *v2, int *res, size_t l) {
    // hipError_t status;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for (; i < l; i+= step) {
	    res[i] = v1[i] + v2[i];
    }

}

int main() {
    int *v1_cpu, *v2_cpu, *res_cpu;

    v1_cpu = calloc(sizeof(int), LEN);
    v2_cpu = calloc(sizeof(int), LEN);

    for (int i = 0; i < LEN; i++) {
	    v1_cpu[i] = i;
	    v2_cpu[i] = i * 40 + 2;
    }

    hipError_t status;
    int *v1_gpu, *v2_gpu, *res_gpu;

    status = hipMalloc((void**)&v1_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&v2_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&res_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipMemcpy(v1_gpu, v1_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMemcpy(v2_gpu, v2_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    free(v1_cpu);
    free(v2_cpu);

    add_vec<<<2, 10, 0>>>(v1_gpu, v2_gpu, res_gpu, LEN);

    res_cpu = calloc(sizeof(int), LEN);
 
    status = hipMemcpy(res_cpu, res_gpu, sizeof(int) * LEN, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipFree(v1_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(v2_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(res_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    free(res_cpu);

    return 0;

}
