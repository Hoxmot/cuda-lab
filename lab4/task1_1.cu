
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using namespace std;

#define LEN 10

__global__ void add_vec(int *v1, int *v2, int *res, size_t l) {
    // cudaError_t status;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for (; i < l; i+= step) {
	    res[i] = v1[i] + v2[i];
    }

}

int main() {
    int v1_cpu[LEN], v2_cpu[LEN], res_cpu[LEN];

    for (int i = 0; i < LEN; i++) {
	    v1_cpu[i] = i;
	    v2_cpu[i] = i * 40 + 2;
    }

    hipError_t status;
    int *v1_gpu, *v2_gpu, *res_gpu;

    status = hipMalloc((void**)&v1_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&v2_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&res_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipMemcpy(v1_gpu, v1_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMemcpy(v2_gpu, v2_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    add_vec<<<2, 10, 0>>>(v1_gpu, v2_gpu, res_gpu, LEN);
 
    status = hipMemcpy(res_cpu, res_gpu, sizeof(int) * LEN, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipFree(v1_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(v2_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(res_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    return 0;

}
