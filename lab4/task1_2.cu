
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

using namespace std;

#define LEN 10

__global__ void add_vec(int *v1, int *v2, int *res, size_t l) {
    // cudaError_t status;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for (; i < l; i+= step) {
	    res[i] = v1[i] + v2[i];
    }

}

__global__ void gen_numbers(int *v1, int *v2, size_t l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for (; i < l; i += step) {
	    v1[i] = i;
	    v2[i] = i * 40 + 2;
    }
}

int main() {

    hipError_t status;
    int *v1_gpu, *v2_gpu, *res_gpu;

    status = hipMalloc((void**)&v1_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&v2_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&res_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    gen_numbers<<<2, 10, 0>>>(v1_gpu, v2_gpu, LEN);

    add_vec<<<2, 10, 0>>>(v1_gpu, v2_gpu, res_gpu, LEN);

    status = hipFree(v1_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(v2_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(res_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    return 0;

}
