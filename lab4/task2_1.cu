
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

using namespace std;

#define LEN 10

__global__ void mul(int *m, int *v, int *res, size_t l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int k;
    for (; i < l * l; i += step) {
	    k = i / l;
	    res[i] = m[i] * v[k];
    }
}

int main() {
    int *m_cpu, *v_cpu, *res_cpu;

    m_cpu = (int*) calloc(LEN * LEN, sizeof(int));
    v_cpu = (int*) calloc(LEN, sizeof(int));

    for (int i = 0; i < LEN; i++) {
	    for (int k = 0; k < LEN; k++) {
	        m_cpu[i * LEN + k] = (i * LEN + k) + 42;
	    }
	    v_cpu[i] = i * 40 + 2;
    }

    for (int i = 0; i < LEN; i++) {
	    for (int k = 0; k < LEN; k++) {
	        cout << m_cpu[i * LEN + k] << " ";
	    }
	    cout << endl;
    }

    cout << endl;

    for (int i = 0; i < LEN; i++) {
	    cout << v_cpu[i] << " ";
    }

    cout << endl;

    hipError_t status;
    int *m_gpu, *v_gpu, *res_gpu;

    status = hipMalloc((void**)&m_gpu, sizeof(int) * LEN * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&v_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&res_gpu, sizeof(int) * LEN * LEN);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipMemcpy(m_gpu, m_cpu, sizeof(int) * LEN * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipMemcpy(v_gpu, v_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    free(m_cpu);
    free(v_cpu);

    mul<<<2, 10, 0>>>(m_gpu, v_gpu, res_gpu, LEN);
 
    res_cpu = (int*) calloc(LEN * LEN, sizeof(int));

    status = hipMemcpy(res_cpu, res_gpu, sizeof(int) * LEN * LEN, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    status = hipFree(m_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(v_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(res_gpu);
    if (status != hipSuccess) {
	    cout << hipGetErrorString(status) << endl;
    }

    for (int i = 0; i < LEN; i++) {
	    for (int k = 0; k < LEN; k++) {
	        cout << res_cpu[i * LEN + k] << " ";
	    }
	    cout << endl;
    }

    free(res_cpu);

    return 0;

}
