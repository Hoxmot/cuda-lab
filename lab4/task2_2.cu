#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>

using namespace std;

#define LEN 10

__global__ void add_vec(int *v1, int *v2, int *res, size_t l) {
    // hipError_t status;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for (; i < l; i+= step) {
	res[i] = v1[i] + v2[i];
    }

}

__global__ void mul(int *m, int *v, int *res, size_t l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int k;
    for (; i < l * l; i += step) {
	k = i / l
	res[i] = m[i] * v[k];
    }
}

int main() {
    int m_cpu[LEN * LEN], v_cpu[LEN], res_cpu[LEN * LEN];

    for (int i = 0; i < LEN; i++) {
	v1_cpu[i] = i;
	v2_cpu[i] = i * 40 + 2;
    }

    hipError_t status;
    int *m_gpu, *v_gpu, *res_gpu;

    status = hipMalloc((void**)&m_gpu, sizeof(int) * LEN * LEN);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&v_gpu, sizeof(int) * LEN);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }
    status = hipMalloc((void**)&res_gpu, sizeof(int) * LEN * LEN);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    status = hipMemcpy(m_gpu, m_cpu, sizeof(int) * LEN * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }
    status = hipMemcpy(v_gpu, v_cpu, sizeof(int) * LEN, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    add_vec<<<2, 10, 0>>>(m_gpu, v_gpu, res_gpu, LEN);
 
    status = hipMemcpy(res_cpu, res_gpu, sizeof(int) * LEN * LEN, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    status = hipFree(v1_gpu);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(v2_gpu);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }
    status = hipFree(res_gpu);
    if (status != hipSuccess) {
	cout << hipGetErrorString(status) << endl;
    }

    for (int i = 0; i < LEN; i++) {
	for (int k = 0; k < LEN; k++) {
	    cout << res_cpu[i * LEN + k] << " ";
	}
	cout << endl;
    }

    return 0;

}
