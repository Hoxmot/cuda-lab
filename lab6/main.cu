#include <hip/hip_runtime_api.h>
#include <stdlib.h>

#include "reduce.h"
#include "limit.h"

#define BLOCK_SIZE 512
#define SIZE 262144

int main () {

    int *data_cpu, *ret_cpu;

    data_cpu = (int*)calloc(SIZE, sizeof(int));

    int *data_gpu, *ret_gpu;
    hipError_t status;

    status = hipMalloc((void**)&data_gpu, SIZE * sizeof(int));
    if (status != hipSuccess) {
	    printf("%s\n", hipGetErrorString(status));
    }

    status = hipMalloc((void**)&ret_gpu, SIZE * sizeof(int));
    if (status != hipSuccess) {
	    printf("%s\n", hipGetErrorString(status));
    }

    status = hipMemcpy(data_gpu, data_cpu, SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    // TODO: call reduce
    reduce<<<SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(data_cpu, ret_cpu);

    status = hipFree(data_gpu);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    status = hipMemcpy(ret_cpu, ret_gpu, sizeof(int), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    status = hipFree(ret_gpu);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    printf("%f\n", ret_cpu);
    free(ret_cpu);
    free(data_cpu);

    return 0;
}