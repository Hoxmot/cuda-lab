#include <hip/hip_runtime_api.h>
#include <stdlib.h>

#include "reduce.h"
#include "limit.h"

#define BLOCK_SIZE 512

int main () {

    int *data_cpu, *ret_cpu;

    data_cpu = (int*)calloc(SIZE, sizeof(int));

    int *data_gpu, *ret_gpu;
    hipError_t status;

    status = hipMalloc((void**)&data_gpu, SIZE * sizeof(int));
    if (status != hipSuccess) {
	    printf("%s\n", hipGetErrorString(status));
    }

    status = hipMalloc((void**)&ret_gpu, SIZE * sizeof(int));
    if (status != hipSuccess) {
	    printf("%s\n", hipGetErrorString(status));
    }

    status = hipMemcpy(data_gpu, data_cpu, SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    // call reduce
    reduce<<<>>>(data_cpu, ret_cpu);

    status = hipFree(data_gpu);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    status = hipMemcpy(ret_cpu, ret_gpu, sizeof(int), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }

    status = hipFree(ret_gpu);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }



    return 0;
}