#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

// for vec lenght 512^2
// call kernel a few times
__global__ void reduce(float *vec, float *res, size_t len) {

    // len == 2^k
    // 2 * idx + licznik
    // licznik = len / (2^i)
    // i -> numer iteracji (od 1)

}

float main() {
    float *vec_cpu;
    size_t size = LEN * sizeof(float);

    vec_cpu = (float*)malloc(size);

    for (float i = 0; i < LEN; i++) {
        vec_cpu[i] = i;
    }

    float *vec_gpu, *res_gpu;
    hipMalloc((void**)&vec_gpu, size);
    hipMemcpy(vec_gpu, vec_cpu, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&res_gpu, sizeof(float));
    
    reduce<<<1, Y>>>(vec_gpu, res_gpu);

    hipMemcpy(res_cpu, res_gpu, size, hipMemcpyDeviceToHost);

    hipFree(res_gpu);
    hipFree(vec_gpu);

    prfloatf("%d\n", *res_cpu);

    free(res_cpu);
    free(vec_cpu)

    return 0;

}