
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>

void handleCudaMalloc(void **var, ssize_t size) {
    hipError_t status;
    status = hipMalloc(var, size);
    if (status != hipSuccess) {
	    printf("%s\n", hipGetErrorString(status));
    }
}

void handleCudaMemcpy(void* dst, const void* src, ssize_t size, hipMemcpyKind kind) {
    hipError_t status;
    status = hipMemcpy(dst, src, size, kind);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }
}

void handleCudaFree(void* pointer) {
    hipError_t status;
    status = hipFree(pointer);
    if (status != hipSuccess) {
        printf("%s\n", hipGetErrorString(status));
    }
}